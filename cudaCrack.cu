#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thread>
#include <chrono>


//Init values
#define INIT_A 0x67452301
#define INIT_B 0xefcdab89
#define INIT_C 0x98badcfe
#define INIT_D 0x10325476

#define SQRT_2 0x5a827999
#define SQRT_3 0x6ed9eba1

__global__ void
NTLM(unsigned int* hash, unsigned int* output, char ch)
{
    unsigned int ident = blockDim.z * blockIdx.z + threadIdx.x;

    // The hash will be prepared before going into cuda.
    unsigned int nt_buffer[16];
    memset(nt_buffer, 0, 16 * 4);
    // hardcoded to hashcat
    // result should be b4b9b02e6f09a9bd760f388b67351e2b
    nt_buffer[0] = (ident%90)+0x20 | ((ident/90)%90)+0x20 << 16;// attempt to bruteforce this value 0x610068;
    ident = ident / (90 * 90);
    nt_buffer[1] = (ident % 90) + 0x20 | ((ident / 90) % 90) + 0x20 << 16; // attempt to brute force this value 0x680073;
    nt_buffer[2] = (blockDim.x % 90) + 0x20 | ((blockDim.x / 90) % 90) + 0x20 << 16;//0x610063;
    int temp = blockDim.x / (90 * 90);
    nt_buffer[3] = (temp % 90) + 0x20 | ((temp / 90) % 90) + 0x20 << 16;//0x800074;
    nt_buffer[14] = 0x70;
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // NTLM hash calculation
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    unsigned int a = INIT_A;
    unsigned int b = INIT_B;
    unsigned int c = INIT_C;
    unsigned int d = INIT_D;

    /* Round 1 */
    a += (d ^ (b & (c ^ d))) + nt_buffer[0]; a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[1]; d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[2]; c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[3]; b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))) + nt_buffer[4]; a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[5]; d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[6]; c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[7]; b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))) + nt_buffer[8]; a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[9]; d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[10]; c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[11]; b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))) + nt_buffer[12]; a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer[13]; d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer[14]; c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer[15]; b = (b << 19) | (b >> 13);

    /* Round 2 */
    a += ((b & (c | d)) | (c & d)) + nt_buffer[0] + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[4] + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[8] + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[12] + SQRT_2; b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer[1] + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[5] + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[9] + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[13] + SQRT_2; b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer[2] + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[6] + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[10] + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[14] + SQRT_2; b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer[3] + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + nt_buffer[7] + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + nt_buffer[11] + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer[15] + SQRT_2; b = (b << 13) | (b >> 19);

    /* Round 3 */
    a += (d ^ c ^ b) + nt_buffer[0] + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[8] + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[4] + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[12] + SQRT_3; b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer[2] + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[10] + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[6] + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[14] + SQRT_3; b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer[1] + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[9] + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[5] + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[13] + SQRT_3; b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer[3] + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + nt_buffer[11] + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + nt_buffer[7] + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer[15] + SQRT_3; b = (b << 15) | (b >> 17);
    output[21] = hash[0];
    output[22] = hash[1];
    output[23] = hash[2] == c + INIT_C;
    output[24] = hash[3] == d + INIT_D;
    if (hash[0] == a + INIT_A &&
        hash[1] == b + INIT_B &&
        hash[2] == c + INIT_C &&
        hash[3] == d + INIT_D) {
        for (int i = 0; i < 16; i++) {
            output[i] = nt_buffer[i];
        }
        output[20] = 1;
    }
    return;
}

__global__ void
NTLM7(unsigned int* hash, unsigned int* output)
{
    // unsigned int ident = 712 * blockIdx.z + threadIdx.x; // blockDim.z wasn't working, so switched to hard coded.
    unsigned int nt_buffer_0 = ((blockIdx.z*(threadIdx.x/90)) % 90) + 0x20 | (blockIdx.z * (threadIdx.x / 90) / 90) + 0x20 << 16;// attempt to bruteforce this value 0x610068;
    unsigned int nt_buffer_1 = (blockIdx.y % 90) + 0x20 | (blockIdx.y / 90) + 0x20 << 16; // attempt to brute force this value 0x680073;
    unsigned int nt_buffer_2 = (blockIdx.x % 90) + 0x20 | (blockIdx.x / 90) + 0x20 << 16; //(blockDim.x % 90) + 0x20 | (blockDim.x / 90) + 0x20 << 16;//0x610063;
    unsigned int nt_buffer_3 = 0x800000 + (threadIdx.x%90)+0x20;// blockDim.y + 0x20;//0x800074;
    unsigned int nt_buffer_14 = 0x70;
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // NTLM hash calculation
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    unsigned int a = INIT_A;
    unsigned int b = INIT_B;
    unsigned int c = INIT_C;
    unsigned int d = INIT_D;

    /* Round 1 */
    a += (d ^ (b & (c ^ d))) + nt_buffer_0; a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))) + nt_buffer_1; d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer_2; c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))) + nt_buffer_3; b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))); a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))); d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))); c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))); b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))); a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))); d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))); c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))); b = (b << 19) | (b >> 13);

    a += (d ^ (b & (c ^ d))); a = (a << 3) | (a >> 29);
    d += (c ^ (a & (b ^ c))); d = (d << 7) | (d >> 25);
    c += (b ^ (d & (a ^ b))) + nt_buffer_14; c = (c << 11) | (c >> 21);
    b += (a ^ (c & (d ^ a))); b = (b << 19) | (b >> 13);

    /* Round 2 */
    a += ((b & (c | d)) | (c & d)) + nt_buffer_0 + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + SQRT_2; b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer_1 + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + SQRT_2; b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer_2 + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + nt_buffer_14 + SQRT_2; b = (b << 13) | (b >> 19);

    a += ((b & (c | d)) | (c & d)) + nt_buffer_3 + SQRT_2; a = (a << 3) | (a >> 29);
    d += ((a & (b | c)) | (b & c)) + SQRT_2; d = (d << 5) | (d >> 27);
    c += ((d & (a | b)) | (a & b)) + SQRT_2; c = (c << 9) | (c >> 23);
    b += ((c & (d | a)) | (d & a)) + SQRT_2; b = (b << 13) | (b >> 19);

    /* Round 3 */
    a += (d ^ c ^ b) + nt_buffer_0 + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + SQRT_3; b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer_2 + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + nt_buffer_14 + SQRT_3; b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer_1 + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + SQRT_3; b = (b << 15) | (b >> 17);

    a += (d ^ c ^ b) + nt_buffer_3 + SQRT_3; a = (a << 3) | (a >> 29);
    d += (c ^ b ^ a) + SQRT_3; d = (d << 9) | (d >> 23);
    c += (b ^ a ^ d) + SQRT_3; c = (c << 11) | (c >> 21);
    b += (a ^ d ^ c) + SQRT_3; b = (b << 15) | (b >> 17);
    if (hash[0] == a &&
        hash[1] == b &&
        hash[2] == c &&
        hash[3] == d) {
        output[0] = nt_buffer_0;
        output[1] = nt_buffer_1;
        output[2] = nt_buffer_2;
        output[3] = nt_buffer_3;
        output[14] = nt_buffer_14;
        output[20] = 1;
    }
    return;
}

unsigned int* getHash(char* hash) {
    unsigned int* h_hash = (unsigned int*)malloc(4 * 4);
    // Verify that allocations succeeded
    if (h_hash == NULL)
    {
        fprintf(stderr, "Failed to allocate host memory!\n");
        exit(EXIT_FAILURE);
    }
    h_hash[0] = 0x2eb0b9b4;
    h_hash[1] = 0xbda9096f;
    h_hash[2] = 0x8b380f76;
    h_hash[3] = 0x2b1e3567;

    h_hash[0] -= INIT_A;
    h_hash[1] -= INIT_B;
    h_hash[2] -= INIT_C;
    h_hash[3] -= INIT_D;
    return h_hash;
}

int debug7Char(void) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(8100,// 90*90
        8100,// 90*90
        737 // 90*90//(1024//90)+1=737
    );
    printf("threads x: %d, y: %d, z: %d\n", threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
    printf("blocks x: %d, y: %d, z: %d\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.z);

    // Allocate the device hash vector
    unsigned int* d_hash = NULL;
    err = hipMalloc((void**)&d_hash, 4 * 4);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device hash (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector
    size_t output_size = 21 * 4;
    unsigned int* output = NULL;
    err = hipMalloc((void**)&output, output_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // allocate host memory
    unsigned int* h_output = (unsigned int*)malloc(output_size);

    // Verify that allocations succeeded
    if ( h_output == NULL)
    {
        fprintf(stderr, "Failed to allocate host memory!\n");
        exit(EXIT_FAILURE);
    }

    unsigned int* h_hash = getHash("b4b9b02e6f09a9bd760f388b67351e2b");

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_hash, h_hash, 4 * 4, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    NTLM7<< <blocksPerGrid, threadsPerBlock >> > (d_hash, output);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("waiting for results");
    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_output, output, output_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("test\n");
    for (int i = 0; i < 16; i++) {
        printf("%#010x", h_output[i]);
        printf(" %c %c\n", ((char*)h_output)[i * 4], ((char*)h_output)[i * 4 + 2]);
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_hash);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(output);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_hash);
    free(h_output);

    printf("Done\n");
    return 0;
}

/**
 * Host main routine
 */
int
main(void)
{

    int dev_count;
    hipGetDeviceCount(&dev_count);
    printf("num devices: %d\n", dev_count);


    hipDeviceProp_t dev_prop;
    for (int i = 0; i < dev_count; i++) {
        hipGetDeviceProperties(&dev_prop, i);
        printf("max threads per block: %d\n", dev_prop.maxThreadsPerBlock);
        printf("max block x dim: %d\n", dev_prop.maxThreadsDim[0]);
        printf("max block y dim: %d\n", dev_prop.maxThreadsDim[1]);
        printf("max block z dim: %d\n", dev_prop.maxThreadsDim[2]);
        printf("max grid x dim: %d\n", dev_prop.maxGridSize[0]);
        printf("max grid y dim: %d\n", dev_prop.maxGridSize[1]);
        printf("max grid z dim: %d\n", dev_prop.maxGridSize[2]);
    }

    return debug7Char();
}

